#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		64		// number of threads per block
#endif

//#ifndef SIZE
//#define SIZE			1*1024*1024	// array size
//#endif

#ifndef NUMTRIALS
#define NUMTRIALS		512000		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif


//helper stuff
float Ranf(float low, float high)
{
	float r = (float)rand();               // 0 - RAND_MAX
	float t = r / (float)RAND_MAX;       // 0. - 1.

	return   low + t * (high - low);
}

int Ranf(int ilow, int ihigh)
{
	float low = (float)ilow;
	float high = ceil((float)ihigh);

	return (int)Ranf(low, high);
}



// ranges for the random numbers:
const float XCMIN = 0.0;
const float XCMAX = 2.0;
const float YCMIN = 0.0;
const float YCMAX = 2.0;
const float RMIN = 0.5;
const float RMAX = 2.0;


// (CUDA Kernel) on the device

__global__  void MonteCarlo( float *xcs, float *ycs, float *rs, float *hits )
{

	//unsigned int numItems = blockDim.x;
	//unsigned int tnum = threadIdx.x;
	//unsigned int wgNum = blockIdx.x;
	//unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;
	
	float xc = xcs[gid];
	float yc = ycs[gid];
	float  r = rs[gid];

	// solve for the intersection using the quadratic formula:
	float a = 2.;
	float b = -2.*(xc + yc);
	float c = xc * xc + yc * yc - r * r;
	float d = b * b - 4.*a*c;

	if (d >= 0.0) {
		// hits the circle:
		// get the first intersection:
		d = sqrt(d);
		float t1 = (-b + d) / (2.*a);	// time to intersect the circle
		float t2 = (-b - d) / (2.*a);	// time to intersect the circle
		float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection

		if (tmin >= 0.0) {
			//Did not engulf laser

			// where does it intersect the circle?
			float xcir = tmin;
			float ycir = tmin;

			// get the unitized normal vector at the point of intersection:
			float nx = xcir - xc;
			float ny = ycir - yc;
			float n = sqrt(nx*nx + ny * ny);
			nx /= n;	// unit vector
			ny /= n;	// unit vector

			// get the unitized incoming vector:
			float inx = xcir - 0.;
			float iny = ycir - 0.;
			float in = sqrt(inx*inx + iny * iny);
			inx /= in;	// unit vector
			iny /= in;	// unit vector

			// get the outgoing (bounced) vector:
			float dot = inx * nx + iny * ny;
			//float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence`
			float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

			// find out if it hits the infinite plate:
			float t = (0. - ycir) / outy;

			if (t >= 0.0) {
				//beam went down
				hits[gid] = 1;
			}
		}
	}
	
}


// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);


	// better to define these here so that the rand() calls don't get into the thread timing:
	float * xcs = new float[NUMTRIALS];
	float * ycs = new float[NUMTRIALS];
	float * rs = new float[NUMTRIALS];
	float * hits = new float[NUMTRIALS];


	// fill the random-value arrays:
	for (int n = 0; n < NUMTRIALS; n++)
	{
		xcs[n] = Ranf(XCMIN, XCMAX);
		ycs[n] = Ranf(YCMIN, YCMAX);
		rs[n] = Ranf(RMIN, RMAX);
		hits[n] = 0;
	}



	// allocate device memory:

	float *d_xcs, *d_ycs, *d_rs, *d_hits;

	dim3 dims_xcs( NUMTRIALS, 1, 1 );
	dim3 dims_ycs( NUMTRIALS, 1, 1 );
	dim3 dims_rs( NUMTRIALS, 1, 1 );
	dim3 dims_hits(NUMTRIALS, 1, 1);

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&d_xcs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&d_ycs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&d_rs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&d_hits), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( d_xcs, xcs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( d_ycs, ycs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( d_rs, rs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( d_hits, hits, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( NUMTRIALS / threads.x, 1, 1 );


	// Create and start timer
	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:


	
	MonteCarlo<<< grid, threads >>>( d_xcs, d_ycs, d_rs, d_hits );
	

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double TrialsPerSecond = (float)NUMTRIALS / secondsTotal;
	double megaTrialsPerSecond = TrialsPerSecond / 1000000.;
	fprintf( stderr, "NUMTRIALS = %10d, BLOCKSIZE = %d, MegaTrials/Second = %10.2lf\n", NUMTRIALS, BLOCKSIZE, megaTrialsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hits, d_hits, ((NUMTRIALS) * sizeof(float)), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// check the sum :

	float prob = 0.0;

	for (int z = 0; z < NUMTRIALS ;z++) {
		if (hits[z] == 1 ) {
			prob = prob + 1;
		}
	}

	prob = prob / NUMTRIALS;

	float probPerc = prob * 100;

	fprintf( stderr, "Probability as percent: %10.2lf\n\n", probPerc);



	// clean up memory:
	delete [ ] xcs;
	delete [ ] ycs;
	delete [ ] rs;
	delete [ ] hits;

	status = hipFree( d_xcs );
		checkCudaErrors( status );
	status = hipFree( d_ycs );
		checkCudaErrors( status );
	status = hipFree( d_rs );
		checkCudaErrors( status );
	status = hipFree(d_hits);
		checkCudaErrors(status);


	return 0;
}





/*

// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			1*1024*1024	// array size
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		100		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// array multiplication (CUDA Kernel) on the device: C = A * B

__global__  void ArrayMul( float *A, float *B, float *C )
{
	__shared__ float prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	prods[tnum] = A[gid] * B[gid];

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		C[wgNum] = prods[0];
}


// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:

	float * hA = new float [ SIZE ];
	float * hB = new float [ SIZE ];
	float * hC = new float [ SIZE/BLOCKSIZE ];

	for( int i = 0; i < SIZE; i++ )
	{
		hA[i] = hB[i] = (float) sqrt(  (float)(i+1)  );
	}

	// allocate device memory:

	float *dA, *dB, *dC;

	dim3 dimsA( SIZE, 1, 1 );
	dim3 dimsB( SIZE, 1, 1 );
	dim3 dimsC( SIZE/BLOCKSIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dA), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dB), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dC), (SIZE/BLOCKSIZE)*sizeof(float) );
		checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dA, hA, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dB, hB, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	for( int t = 0; t < NUMTRIALS; t++)
	{
	        ArrayMul<<< grid, threads >>>( dA, dB, dC );
	}

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)SIZE * (float)NUMTRIALS / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	fprintf( stderr, "Array Size = %10d, MegaMultReductions/Second = %10.2lf\n", SIZE, megaMultsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hC, dC, (SIZE/BLOCKSIZE)*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// check the sum :

	double sum = 0.;
	for(int i = 0; i < SIZE/BLOCKSIZE; i++ )
	{
		//fprintf(stderr, "hC[%6d] = %10.2f\n", i, hC[i]);
		sum += (double)hC[i];
	}
	fprintf( stderr, "\nsum = %10.2lf\n", sum );

	// clean up memory:
	delete [ ] hA;
	delete [ ] hB;
	delete [ ] hC;

	status = hipFree( dA );
		checkCudaErrors( status );
	status = hipFree( dB );
		checkCudaErrors( status );
	status = hipFree( dC );
		checkCudaErrors( status );


	return 0;
}

*/